
#include <hip/hip_runtime.h>
__global__ void insertionSortNetwork(float* x, int N) {
    extern __shared__ float y[];
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    float res = x[idx];
    __syncthreads();
    x[idx] = blockIdx.x;
    /*float res = 0.0;
    for (int i = 0; i < 2*N-3; i++) {
        int j = 2*N-4-i;
        if (idx < N) {
            res = x[idx];
            if ((i <  N-1 && idx < i+2) || 
                (i >= N-1 && idx < j+2)) {
                if (i%2 == 0) {
                    if (idx %2 == 0) {
                        res = x[idx];
                        if (x[idx+1] < x[idx]) {
                            res = x[idx+1];
                        }
                    }
                    else {
                        res = x[idx];
                        if (x[idx-1] > x[idx]) {
                            res = x[idx-1];
                        }
                    }
                }
                else if (idx > 0) {
                    if (idx %2 == 0) {
                        res = x[idx];
                        if (x[idx-1] > x[idx]) {
                            res = x[idx-1];
                        }
                    }
                    else {
                        res = x[idx];
                        if (x[idx+1] < x[idx]) {
                            res = x[idx+1];
                        }
                    }
                }
            }
        }
        __syncthreads();
        if (idx < N) {
            x[idx] = res;
        }
        __syncthreads();
    }*/
}

